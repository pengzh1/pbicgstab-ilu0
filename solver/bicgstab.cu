#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <stdio.h>
#include <vector>
#include <time.h>
#include <iomanip>
#include <thread>
#include <string>
#include <sstream>
#include "ilu.cuh"

#ifndef WARP_SIZE
#define WARP_SIZE   32
#endif

#ifndef WARP_PER_BLOCK
#define WARP_PER_BLOCK  32
#endif

void cudaCheckError2() {
    hipDeviceSynchronize();
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        std::stringstream _error;
        _error << "Cuda failure: '" << hipGetErrorString(e) << "'";
        throw 999;
    }
}

void checkCudaError(hipError_t e) {
    if (e != hipSuccess) {
        std::stringstream _error;
        _error << "Cuda failure: '" << hipGetErrorString(e) << "'";
        hipDeviceSynchronize();
        throw "xxx";
    }
}

void setUpDescriptor(hipsparseMatDescr_t &descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, matrixType);
    hipsparseSetMatIndexBase(descrA, indexBase);
}

__global__ void printArrF(const double *val) {
    printf("data is ");
    for (int i = 0; i < 20; i++) {
        printf("%f ", val[i]);
    }
    printf("\n");
}

__global__ void printArrI(const int *val) {
    printf("dataI is ");
    for (int i = 0; i < 20; i++) {
        printf("%d ", val[i]);
    }
    printf("\n");
}

__global__
void spTrSolveL(const int *__restrict__ d_csrRowPtr,
                const int *__restrict__ d_csrColIdx,
                const double *__restrict__ d_csrVal,
                volatile bool *__restrict__ d_get_value,// 0*m
                const int m, // rows
                const double *__restrict__ d_b, // rhs
                double *d_x, // initVec
                int *d_id_extractor // 0
) {
//    const int global_id = atomicAdd(d_id_extractor, 1);
    const int global_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_id >= m) {
        return;
    }
    int col, j, i;
    double xi;
    double left_sum = 0;
    i = global_id; // 3
    j = d_csrRowPtr[i];
    if (d_csrColIdx[j] > i) {
        return;
    }
    int end = d_csrRowPtr[i + 1] - 1;
    while (d_csrColIdx[end] > i) {
        end -= 1;
    }
    end += 1;
    while (j < end) { // 1,2
        col = d_csrColIdx[j];
        while (d_get_value[col] && j < end) {
            left_sum += d_csrVal[j] * d_x[col];
            j++;
            if (j < end) {
                col = d_csrColIdx[j];
            }
        }
        if (i == col || j == end) {
            xi = (d_b[i] - left_sum);
            d_x[i] = xi;
            __threadfence();
            d_get_value[i] = true;
            j++;
        }
    }
}

__global__
void spTrSolveU(const int *__restrict__ d_csrRowPtr,
                const int *__restrict__ d_csrColIdx,
                const double *__restrict__ d_csrVal,
                volatile bool *__restrict__ d_get_value,// 0*m
                const int m, // rows
                const double *__restrict__ d_b, // rhs
                double *d_x, // initVec
                int *d_id_extractor // 0
) {
//    int global_idx = atomicAdd(d_id_extractor, 1);
    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;
    const int global_id = m - global_idx - 1;
    if (global_id < 0) {
        return;
    }
    int col, j, i;
    col = -1;
    double xi;
    double right_sum = 0;
    i = global_id; // 3
    j = d_csrRowPtr[i + 1] - 1;
    if(d_csrColIdx[j] < i) {
        return;
    }
    int end = d_csrRowPtr[i];
    while (d_csrColIdx[end] < i) {
        end += 1;
    }
    int itr = 0;
    while (j >= end && d_csrColIdx[j] >= i) { // 1,2
        itr += 1;
        col = d_csrColIdx[j];
        while (j >= end && d_csrColIdx[j] > i && d_get_value[col]) {
            right_sum += d_csrVal[j] * d_x[col];
            j--;
            if (j >= end && d_csrColIdx[j] > i) {
                col = d_csrColIdx[j];
            }
        }
        if (i == col || j == end) {
            xi = (d_b[i] - right_sum) / (d_csrVal[j]);
            d_x[i] = xi;
            __threadfence();
            d_get_value[i] = true;
            j--;
        }
    }
}


int spTrSolve(const int *__restrict__ d_csrRowPtr,
              const int *__restrict__ d_csrColIdx,
              const double *__restrict__ d_csrVal,
              const int m, // rows
              const int nnz, // nnz for L
              const double *__restrict__ d_b, // rhs
              double *d_x,// initVec,
              bool isL
) {

    //get_value
    bool *d_get_value;
    hipMalloc((void **) &d_get_value, (m) * sizeof(bool));
    hipMemset(d_get_value, false, sizeof(bool) * m);
    // step 5: solve L*y = x
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;;
    int num_blocks = ceil((double) m / (double) (num_threads));
    int *d_id_extractor;
    hipMalloc((void **) &d_id_extractor, sizeof(int));
    hipMemset(d_x, 0, sizeof(double) * m);
    hipMemset(d_id_extractor, 0, sizeof(int));
    if (isL) {
        spTrSolveL<<< num_blocks, num_threads >>>
                (d_csrRowPtr, d_csrColIdx, d_csrVal,
                 d_get_value, m, d_b, d_x, d_id_extractor);
    } else {
        spTrSolveU<<< num_blocks, num_threads >>>
                (d_csrRowPtr, d_csrColIdx, d_csrVal,
                 d_get_value, m, d_b, d_x, d_id_extractor);
    }
    cudaCheckError2();
    hipDeviceSynchronize();


    return 0;

}

void setDesc(hipsparseMatDescr_t &descrLU, hipsparseMatrixType_t matrixType,
             hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode,
             hipsparseDiagType_t diagType) {
    hipsparseCreateMatDescr(&descrLU);
    hipsparseSetMatType(descrLU, matrixType);
    hipsparseSetMatIndexBase(descrLU, indexBase);
    hipsparseSetMatFillMode(descrLU, fillMode);
    hipsparseSetMatDiagType(descrLU, diagType);
}


void checkSpError(hipsparseStatus_t error) {
    hipDeviceSynchronize();
    switch (error) {
        case HIPSPARSE_STATUS_SUCCESS:
            break;
        default:
            throw "Sp ERROR";
    }
}

void memQuery(csrilu02Info_t &infoA, csrsv2Info_t &infoL, csrsv2Info_t &infoU,
              hipsparseHandle_t cusparseHandle, const int n, const int nnz,
              hipsparseMatDescr_t &descrA, hipsparseMatDescr_t &descrL, hipsparseMatDescr_t &descrU,
              double *d_A, const int *d_A_RowPtr, const int *d_A_ColInd,
              hipsparseOperation_t matrixOperation, void **pBuffer) {
    hipsparseCreateCsrilu02Info(&infoA);
    hipsparseCreateCsrsv2Info(&infoL);
    hipsparseCreateCsrsv2Info(&infoU);

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
    hipsparseDcsrilu02_bufferSize(cusparseHandle, n, nnz, descrA, d_A, d_A_RowPtr,
                                 d_A_ColInd, infoA, &pBufferSize_M);
    hipsparseDcsrsv2_bufferSize(cusparseHandle, matrixOperation, n, nnz, descrL,
                               d_A, d_A_RowPtr, d_A_ColInd, infoL, &pBufferSize_L);
    hipsparseDcsrsv2_bufferSize(cusparseHandle, matrixOperation, n, nnz, descrU,
                               d_A, d_A_RowPtr, d_A_ColInd, infoU, &pBufferSize_U);

    int pBufferSize = std::max(pBufferSize_M, std::max(pBufferSize_L, pBufferSize_U));

    checkCudaError(hipMalloc((void **) pBuffer, pBufferSize));
}


void spAnalyze(csrilu02Info_t &infoA, csrsv2Info_t &infoL,
               csrsv2Info_t &infoU, hipsparseHandle_t cusparseHandle, const int N,
               const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t &descrL,
               hipsparseMatDescr_t &descrU, double *d_A, const int *d_A_RowPtr,
               const int *d_A_ColInd, hipsparseOperation_t matrixOperation,
               hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2,
               void *pBuffer) {
    int structural_zero;
    time_t t1 = clock();


    checkSpError(hipsparseDcsrilu02_analysis(cusparseHandle, N, nnz, descrA, d_A, d_A_RowPtr,
                                            d_A_ColInd, infoA, solvePolicy1, pBuffer));

    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoA, &structural_zero);

    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        printf("A(%d, %d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseDcsrsv2_analysis(cusparseHandle, matrixOperation, N, nnz, descrL,
                             d_A, d_A_RowPtr, d_A_ColInd, infoL, solvePolicy1, pBuffer);
    hipsparseDcsrsv2_analysis(cusparseHandle, matrixOperation, N, nnz, descrU,
                             d_A, d_A_RowPtr, d_A_ColInd, infoU, solvePolicy2, pBuffer);
    hipDeviceSynchronize();
//    printf("luCost%ld", (clock() - t1) / (CLOCKS_PER_SEC / 1000));
}

__global__ void printStrF() {

}

void setUpMatrix(hipsparseHandle_t &cusparseHandle, hipsparseMatDescr_t &descrA,
                 hipsparseMatDescr_t &descrL, hipsparseMatDescr_t &descrU, csrilu02Info_t &infoA,
                 csrsv2Info_t &infoL, csrsv2Info_t &infoU, int n, int nnz, double *valACopy,
                 const int *rowPtr, const int *colInd, void **pBuffer) {
    setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO);
    setDesc(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO,
            HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
    setDesc(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO,
            HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // Step 2: Query how much memory used in LU factorization and the two following system inversions.
    memQuery(infoA, infoL, infoU, cusparseHandle, n, nnz, descrA, descrL, descrU,
             valACopy, rowPtr, colInd, HIPSPARSE_OPERATION_NON_TRANSPOSE, pBuffer);
//     Step 3: Analyze the three problems: LU factorization and the two following system inversions.
//    spAnalyze(infoA, infoL, infoU, cusparseHandle, n, nnz, descrA, descrL, descrU,
//              valACopy, rowPtr, colInd, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, *pBuffer);
    cudaCheckError2();
}

// TODO perfFix
void spNewMV(hipsparseHandle_t handle,
             hipsparseOperation_t transA,
             int m,
             int n,
             int nnz,
             const double *alpha,
             const hipsparseMatDescr_t descrA,
             const double *csrValA,
             const int *csrRowPtrA,
             const int *csrColIndA,
             const double *x,
             const double *beta,
             double *y) {
    time_t mv0 = clock();
    double one = 1, nega_one = -1, zero = 0;
    int *rows = const_cast<int *>(csrRowPtrA);
    int *cols = const_cast<int *>(csrColIndA);
    double *vals = const_cast<double *>(csrValA);
    hipsparseSpMatDescr_t matA_descr;
    hipsparseDnVecDescr_t vecX_descr;
    hipsparseDnVecDescr_t vecY_descr;
    time_t mv1 = clock();
    checkSpError(hipsparseCreateDnVec(&vecX_descr, m, const_cast<double *>(x), HIP_R_64F));
    checkSpError(hipsparseCreateDnVec(&vecY_descr, m, const_cast<double *>(y), HIP_R_64F));
    checkSpError(hipsparseCreateCsr(&matA_descr, m, n, nnz, const_cast<int *>(rows), const_cast<int *>(cols),
                                   const_cast<double *>(vals),
                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    size_t bufferSize = 0;
    checkSpError(hipsparseSpMV_bufferSize(handle, transA, alpha, matA_descr, vecX_descr, beta, vecY_descr, HIP_R_64F,
                                         HIPSPARSE_CSRMV_ALG1, &bufferSize));
    cudaCheckError2();
    void *dBuffer = NULL;
    hipMalloc(&dBuffer, bufferSize);
    hipDeviceSynchronize();
    time_t mv2 = clock();
    checkSpError(hipsparseSpMV(handle, transA, alpha, matA_descr, vecX_descr, beta, vecY_descr, HIP_R_64F,
                              HIPSPARSE_SPMV_CSR_ALG2, dBuffer));
    hipDeviceSynchronize();
    cudaCheckError2();
    time_t mv3 = clock();
    hipsparseDestroySpMat(matA_descr);
    hipsparseDestroyDnVec(vecX_descr);
    hipsparseDestroyDnVec(vecY_descr);
    hipDeviceSynchronize();
    cudaCheckError2();
    time_t mv4 = clock();
//    printf("mvTime %ld %ld %ld %ld \n",
//           (mv4 - mv3) / (CLOCKS_PER_SEC / 1000), (mv3 - mv2) / (CLOCKS_PER_SEC / 1000),
//           (mv2 - mv1) / (CLOCKS_PER_SEC / 1000),
//           (mv1 - mv0) / (CLOCKS_PER_SEC / 1000));
}

void spSolverBiCGStab(int n, int nnz, const double *valA, const int *rowPtr, const int *colInd,
                      const double *b, double *x, double tol, hipsparseHandle_t cusparseHandle,
                      hipblasHandle_t cublasHandle) {
    time_t solve_start = clock();
    // Create descriptors for A, L and U.
    hipsparseMatDescr_t descrA, descrL, descrU;

    // Create ILU and SV info for A, L and U.
    csrilu02Info_t infoA;
    csrsv2Info_t infoL, infoU;

    // Create a copy of A for incomplete LU decomposition.
    // This copy will be modified in the solving process.
    double *valACopy;
    hipMalloc((void **) &valACopy, nnz * sizeof(double));
    hipMemcpy(valACopy, valA, nnz * sizeof(double), hipMemcpyDeviceToDevice);

    // Incomplete LU.
    time_t solve_start1 = clock();
    void *pBuffer;
    setUpMatrix(cusparseHandle, descrA, descrL, descrU, infoA, infoL, infoU, n, nnz, valACopy, rowPtr, colInd,
                &pBuffer);
    int *diag_info = nullptr;
    hipMalloc((void **) &diag_info, sizeof(int) * n);
    cudaCheckError2();
    ILU0_MEGA(rowPtr, colInd, valACopy,
              n,
              nnz);
    hipDeviceSynchronize();
    time_t solve_start2 = clock();
    double *r;
    checkCudaError(hipMalloc((void **) &r, n * sizeof(double)));
    double *rw;
    hipMalloc((void **) &rw, n * sizeof(double));
    double *p;
    hipMalloc((void **) &p, n * sizeof(double));
    double *ph;
    hipMalloc((void **) &ph, n * sizeof(double));
    double *t;
    hipMalloc((void **) &t, n * sizeof(double));
    double *q;
    hipMalloc((void **) &q, n * sizeof(double));
    double *s;
    checkCudaError(hipMalloc((void **) &s, n * sizeof(double)));
    time_t solve_start3 = clock();

    double one = 1, nega_one = -1, zero = 0;
    double alpha, negalpha, beta, omega, nega_omega;
    double temp1, temp2;
    double rho = 0.0, rhop;
    double nrmr0;
    double nrmr;
    int niter = 0;

    // Initial guess x0 (all zeros here).
//    hipblasDscal(cublasHandle, n, &zero, x, 1);
    // 1: compute the initial residual r = b - A * x0.
    spNewMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &nega_one, descrA, valA, rowPtr,
            colInd, x, &zero, r);
    cudaCheckError2();

    hipblasDaxpy(cublasHandle, n, &one, b, 1, r, 1);
    // 2: copy r into rw and p.
    hipblasDcopy(cublasHandle, n, r, 1, rw, 1);
    hipblasDcopy(cublasHandle, n, r, 1, p, 1);
    time_t solve_start4 = clock();
    hipblasDnrm2(cublasHandle, n, r, 1, &nrmr0);
    printf("initNRMR %f \n", nrmr0);
    // Repeat until convergence.
    while (true) {
        printf("niter %d ms %ld\n", niter, clock() / (CLOCKS_PER_SEC / 1000));
        time_t it0 = clock();
        rhop = rho;
        hipblasDdot(cublasHandle, n, rw, 1, r, 1, &rho);
        if (niter > 0) {
            beta = (rho / rhop) * (alpha / omega);
            //  p = r + beta * (p - omega * v)
            hipblasDaxpy(cublasHandle, n, &nega_omega, q, 1, p, 1);  // p += -omega * v
            hipblasDscal(cublasHandle, n, &beta, p, 1);  // p *= beta
            hipblasDaxpy(cublasHandle, n, &one, r, 1, p, 1);  // p += 1 * r
        }
        hipDeviceSynchronize();
        time_t it1 = clock();
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, p, t, true);
        hipDeviceSynchronize();
        time_t it2 = clock();
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, t, ph, false);
        spNewMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, valA, rowPtr, colInd,
                ph, &zero, q);
        hipDeviceSynchronize();
        time_t it3 = clock();
        hipblasDdot(cublasHandle, n, rw, 1, q, 1, &temp1);
        alpha = rho / temp1;
        negalpha = -alpha;
        hipblasDaxpy(cublasHandle, n, &negalpha, q, 1, r, 1);
        hipblasDaxpy(cublasHandle, n, &alpha, ph, 1, x, 1);
        hipblasDnrm2(cublasHandle, n, r, 1, &nrmr);
        hipDeviceSynchronize();
        time_t it4 = clock();
        if ((nrmr / nrmr0) < tol) {
            std::cout << std::setprecision(12) << nrmr / nrmr0 << " " << nrmr << " NRMR \n";
            break;
        }
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, r, t, true);
        hipDeviceSynchronize();
        time_t it5 = clock();
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, t, s, false);
        spNewMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, valA, rowPtr, colInd,
                s, &zero, t);
        hipblasDdot(cublasHandle, n, t, 1, r, 1, &temp1);
        hipDeviceSynchronize();
        time_t it6 = clock();
        hipblasDdot(cublasHandle, n, t, 1, t, 1, &temp2);
        omega = temp1 / temp2;
        nega_omega = -omega;
        hipblasDaxpy(cublasHandle, n, &omega, s, 1, x, 1);
        hipblasDaxpy(cublasHandle, n, &nega_omega, t, 1, r, 1);
        hipblasDnrm2(cublasHandle, n, r, 1, &nrmr);
        hipDeviceSynchronize();
        time_t it7 = clock();
        printf("itTime %ld %ld %ld %ld %ld %ld %ld \n", (it7 - it6) / (CLOCKS_PER_SEC / 1000),
               (it6 - it5) / (CLOCKS_PER_SEC / 1000),
               (it5 - it4) / (CLOCKS_PER_SEC / 1000),
               (it4 - it3) / (CLOCKS_PER_SEC / 1000),
               (it3 - it2) / (CLOCKS_PER_SEC / 1000),
               (it2 - it1) / (CLOCKS_PER_SEC / 1000),
               (it1 - it0) / (CLOCKS_PER_SEC / 1000));
        if ((nrmr / nrmr0) < tol) {
            std::cout << std::setprecision(12) << nrmr / nrmr0 << nrmr << " NRMR \n";
            break;
        }
        niter++;
    }
    time_t solve_start5 = clock();
    time_t solve_start6 = clock();
    time_t solve_end = clock();
    printf("solveTime %ld %ld %ld %ld %ld %ld %ld %ld\n",
           (solve_end - solve_start) / (CLOCKS_PER_SEC / 1000),
           (solve_end - solve_start6) / (CLOCKS_PER_SEC / 1000),
           (solve_start6 - solve_start5) / (CLOCKS_PER_SEC / 1000),
           (solve_start5 - solve_start4) / (CLOCKS_PER_SEC / 1000),
           (solve_start4 - solve_start3) / (CLOCKS_PER_SEC / 1000),
           (solve_start3 - solve_start2) / (CLOCKS_PER_SEC / 1000),
           (solve_start2 - solve_start1) / (CLOCKS_PER_SEC / 1000),
           (solve_start1 - solve_start) / (CLOCKS_PER_SEC / 1000));
}
