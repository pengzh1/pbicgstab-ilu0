#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <stdio.h>
#include <vector>
#include <time.h>
#include <iomanip>
#include <thread>
#include <string>
#include <sstream>
#include "ilu.cuh"

#ifndef WARP_SIZE
#define WARP_SIZE   32
#endif

#ifndef WARP_PER_BLOCK
#define WARP_PER_BLOCK  32
#endif

void cudaCheckError2() {
    hipDeviceSynchronize();
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        std::stringstream _error;
        _error << "Cuda failure: '" << hipGetErrorString(e) << "'";
        throw 999;
    }
}

void checkCudaError(hipError_t e) {
    if (e != hipSuccess) {
        std::stringstream _error;
        _error << "Cuda failure: '" << hipGetErrorString(e) << "'";
        hipDeviceSynchronize();
        throw "xxx";
    }
}
void setUpDescriptor(hipsparseMatDescr_t &descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, matrixType);
    hipsparseSetMatIndexBase(descrA, indexBase);
}

__global__ void printArrF(const double *val) {
    printf("data is ");
    for (int i = 0; i < 20; i++) {
        printf("%f ", val[i]);
    }
    printf("\n");
}
__global__ void printArrI(const int *val) {
    printf("dataI is ");
    for (int i = 0; i < 20; i++) {
        printf("%d ", val[i]);
    }
    printf("\n");
}

__global__
void spTrSolveL(const int *__restrict__ d_csrRowPtr,
                const int *__restrict__ d_csrColIdx,
                const double *__restrict__ d_csrVal,
                volatile int *d_get_value,// 0*m
                const int m, // rows
                const int nnz, // nnz for L
                const double *__restrict__ d_b, // rhs
                double *d_x, // initVec
                const int begin,// 0
                int *d_id_extractor // 0
) {
    const int global_id = atomicAdd(d_id_extractor, 1);
    if (global_id >= m || d_get_value[global_id] == 1) {
        return;
    }
    int col, j, i;
    col = -1;
    double xi;
    double left_sum = 0;
    i = global_id; // 3
    j = d_csrRowPtr[i];
    int itr = 0;
    // 整行无元素，直接求解
    if (d_csrRowPtr[i + 1] == d_csrRowPtr[i]) {
        xi = (d_b[i] - left_sum);
        d_x[i] = xi;
        __threadfence();
        d_get_value[i] = 1;
        printf("beginx %d %d %f\n", global_id, i, d_x[i]);
    }
//    __syncthreads();
    itr = 0;
//    __threadfence();
    while (j < d_csrRowPtr[i + 1] && d_csrColIdx[j] <= i) { // 1,2
        itr += 1;
        col = d_csrColIdx[j];
        while (d_get_value[col] == 1 && j < d_csrRowPtr[i + 1] && d_csrColIdx[j] < i) {
            left_sum += d_csrVal[j] * d_x[col];
            j++;
            if (j < d_csrRowPtr[i + 1] && d_csrColIdx[j] < i) {
                col = d_csrColIdx[j];
            }
        }
        if (i == col || j == d_csrRowPtr[i + 1]) {
            xi = (d_b[i] - left_sum);
            d_x[i] = xi;
            __threadfence();
            d_get_value[i] = 1;
            j++;
        }
    }
}

__global__
void spTrSolveU(const int *__restrict__ d_csrRowPtr,
                const int *__restrict__ d_csrColIdx,
                const double *__restrict__ d_csrVal,
                volatile int *d_get_value,// 0*m
                const int m, // rows
                const int nnz, // nnz for L
                const double *__restrict__ d_b, // rhs
                double *d_x, // initVec
                const int begin,// 0
                int *d_id_extractor // 0
) {
    int global_idx = atomicAdd(d_id_extractor, 1);
    const int global_id = m - global_idx - 1;
    int adj = 0;
    if (global_id < 0 || d_get_value[global_id] == 1) {
        return;
    }
    int col, j, i;
    col = -1;
    double xi;
    double right_sum = 0;
    i = global_id; // 3

    j = d_csrRowPtr[i + 1] - 1;
    int itr = 0;
    __syncthreads();
    __threadfence();
    while (j >= d_csrRowPtr[i] && d_csrColIdx[j] - adj >= i) { // 1,2
        itr += 1;
        col = d_csrColIdx[j];
        if (itr % 100000 == 0 && global_id < 200000) {
            printf("add2 nPtr%d glid%d j%d col%d sum%f geted%d geted%d wait%d \n ", d_csrRowPtr[i + 1], global_id, j,
                   col,
                   right_sum, d_get_value[i], d_get_value[col - adj], col - adj);
        }
        while (j >= d_csrRowPtr[i] && d_csrColIdx[j] - adj > i && d_get_value[col - adj] == 1) {
            right_sum += d_csrVal[j] * d_x[col - adj];
            j--;
            if (j >= d_csrRowPtr[i] && d_csrColIdx[j] - adj > i) {
                col = d_csrColIdx[j];
            }
        }
        if (i == col - adj || j == d_csrRowPtr[i]) {
            xi = (d_b[i] - right_sum) / (d_csrVal[j]);
            d_x[i] = xi;
            __threadfence();
            d_get_value[i] = 1;
            __threadfence();
            j--;
        }
        __threadfence();
    }
    __threadfence();
}


int spTrSolve(const int *__restrict__ d_csrRowPtr,
              const int *__restrict__ d_csrColIdx,
              const double *__restrict__ d_csrVal,
              const int m, // rows
              const int nnz, // nnz for L
              const double *__restrict__ d_b, // rhs
              double *d_x,// initVec,
              bool isL
) {


    int *d_get_value;
    //get_value
    int *get_value = (int *) malloc(m * sizeof(int));
    memset(get_value, 0, m * sizeof(int));
    hipMalloc((void **) &d_get_value, (m) * sizeof(int));
    hipMemcpy(d_get_value, get_value, (m) * sizeof(int), hipMemcpyHostToDevice);
    // step 5: solve L*y = x
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;;
    int num_blocks = ceil((double) m / (double) (num_threads));
    int *d_id_extractor;
    hipMalloc((void **) &d_id_extractor, sizeof(int));
    hipMemset(d_get_value, 0, sizeof(int) * m);
    hipMemset(d_x, 0, sizeof(double) * m);
    hipMemset(d_id_extractor, 0, sizeof(int));
    if (isL) {
        spTrSolveL<<< num_blocks, num_threads >>>
                (d_csrRowPtr, d_csrColIdx, d_csrVal,
                 d_get_value, m, nnz, d_b, d_x, 0, d_id_extractor);
    } else {
        spTrSolveU<<< num_blocks, num_threads >>>
                (d_csrRowPtr, d_csrColIdx, d_csrVal,
                 d_get_value, m, nnz, d_b, d_x, 0, d_id_extractor);
    }
    cudaCheckError2();
    hipDeviceSynchronize();


    return 0;

}

/// Set up descriptor for LU.
void setUpDescriptorLU(hipsparseMatDescr_t &descrLU, hipsparseMatrixType_t matrixType,
                       hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode,
                       hipsparseDiagType_t diagType) {
    hipsparseCreateMatDescr(&descrLU);
    hipsparseSetMatType(descrLU, matrixType);
    hipsparseSetMatIndexBase(descrLU, indexBase);
    hipsparseSetMatFillMode(descrLU, fillMode);
    hipsparseSetMatDiagType(descrLU, diagType);
}


void checkSpError(hipsparseStatus_t error) {
    hipDeviceSynchronize();
    switch (error) {
        case HIPSPARSE_STATUS_SUCCESS:
            break;
        default:
            throw "Sp ERROR";
    }
}

/// Memory query for LU.
void memoryQueryLU(csrilu02Info_t &infoA, csrsv2Info_t &infoL, csrsv2Info_t &infoU,
                   hipsparseHandle_t cusparseHandle, const int n, const int nnz,
                   hipsparseMatDescr_t &descrA, hipsparseMatDescr_t &descrL, hipsparseMatDescr_t &descrU,
                   double *d_A, const int *d_A_RowPtr, const int *d_A_ColInd,
                   hipsparseOperation_t matrixOperation, void **pBuffer) {
    hipsparseCreateCsrilu02Info(&infoA);
    hipsparseCreateCsrsv2Info(&infoL);
    hipsparseCreateCsrsv2Info(&infoU);

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
    hipsparseDcsrilu02_bufferSize(cusparseHandle, n, nnz, descrA, d_A, d_A_RowPtr,
                                 d_A_ColInd, infoA, &pBufferSize_M);
    hipsparseDcsrsv2_bufferSize(cusparseHandle, matrixOperation, n, nnz, descrL,
                               d_A, d_A_RowPtr, d_A_ColInd, infoL, &pBufferSize_L);
    hipsparseDcsrsv2_bufferSize(cusparseHandle, matrixOperation, n, nnz, descrU,
                               d_A, d_A_RowPtr, d_A_ColInd, infoU, &pBufferSize_U);

    int pBufferSize = std::max(pBufferSize_M, std::max(pBufferSize_L, pBufferSize_U));

    checkCudaError(hipMalloc((void **) pBuffer, pBufferSize));
}

/// Analysis for LU.
void analyzeLU(csrilu02Info_t &infoA, csrsv2Info_t &infoL,
               csrsv2Info_t &infoU, hipsparseHandle_t cusparseHandle, const int N,
               const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t &descrL,
               hipsparseMatDescr_t &descrU, double *d_A, const int *d_A_RowPtr,
               const int *d_A_ColInd, hipsparseOperation_t matrixOperation,
               hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2,
               void *pBuffer) {
    int structural_zero;
    time_t t1 = clock();


    checkSpError(hipsparseDcsrilu02_analysis(cusparseHandle, N, nnz, descrA, d_A, d_A_RowPtr,
                                            d_A_ColInd, infoA, solvePolicy1, pBuffer));

    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoA, &structural_zero);

    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        printf("A(%d, %d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseDcsrsv2_analysis(cusparseHandle, matrixOperation, N, nnz, descrL,
                             d_A, d_A_RowPtr, d_A_ColInd, infoL, solvePolicy1, pBuffer);
    hipsparseDcsrsv2_analysis(cusparseHandle, matrixOperation, N, nnz, descrU,
                             d_A, d_A_RowPtr, d_A_ColInd, infoU, solvePolicy2, pBuffer);
    hipDeviceSynchronize();
    printf("luCost%ld", (clock() - t1) / (CLOCKS_PER_SEC / 1000));
}

__global__ void printStrF() {

}

void setUpMatrix(hipsparseHandle_t &cusparseHandle, hipsparseMatDescr_t &descrA,
                 hipsparseMatDescr_t &descrL, hipsparseMatDescr_t &descrU, csrilu02Info_t &infoA,
                 csrsv2Info_t &infoL, csrsv2Info_t &infoU, int n, int nnz, double *valACopy,
                 const int *rowPtr, const int *colInd, void **pBuffer) {
    setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO);
    setUpDescriptorLU(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO,
                      HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
    setUpDescriptorLU(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO,
                      HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // Step 2: Query how much memory used in LU factorization and the two following system inversions.
    memoryQueryLU(infoA, infoL, infoU, cusparseHandle, n, nnz, descrA, descrL, descrU,
                  valACopy, rowPtr, colInd, HIPSPARSE_OPERATION_NON_TRANSPOSE, pBuffer);
//     Step 3: Analyze the three problems: LU factorization and the two following system inversions.
    analyzeLU(infoA, infoL, infoU, cusparseHandle, n, nnz, descrA, descrL, descrU,
              valACopy, rowPtr, colInd, HIPSPARSE_OPERATION_NON_TRANSPOSE,
              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, *pBuffer);
//     Step 4: Factorize A = L * U (A will be overwritten).
//    computeLU(infoA, cusparseHandle, n, nnz, descrA, valACopy, rowPtr, colInd,
//              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, *pBuffer);
    cudaCheckError2();
}

// TODO perfFix
void spNewMV(hipsparseHandle_t handle,
             hipsparseOperation_t transA,
             int m,
             int n,
             int nnz,
             const double *alpha,
             const hipsparseMatDescr_t descrA,
             const double *csrValA,
             const int *csrRowPtrA,
             const int *csrColIndA,
             const double *x,
             const double *beta,
             double *y) {
    time_t mv0 = clock();
    double one = 1, nega_one = -1, zero = 0;
    int *rows = const_cast<int *>(csrRowPtrA);
    int *cols = const_cast<int *>(csrColIndA);
    double *vals = const_cast<double *>(csrValA);
    hipsparseSpMatDescr_t matA_descr;
    hipsparseDnVecDescr_t vecX_descr;
    hipsparseDnVecDescr_t vecY_descr;
    time_t mv1 = clock();
    checkSpError(hipsparseCreateDnVec(&vecX_descr, m, const_cast<double *>(x), HIP_R_64F));
    checkSpError(hipsparseCreateDnVec(&vecY_descr, m, const_cast<double *>(y), HIP_R_64F));
    checkSpError(hipsparseCreateCsr(&matA_descr, m, n, nnz, const_cast<int *>(rows), const_cast<int *>(cols),
                                   const_cast<double *>(vals),
                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    size_t bufferSize = 0;
    checkSpError(hipsparseSpMV_bufferSize(handle, transA, alpha, matA_descr, vecX_descr, beta, vecY_descr, HIP_R_64F,
                                         HIPSPARSE_CSRMV_ALG1, &bufferSize));
    cudaCheckError2();
    void *dBuffer = NULL;
    hipMalloc(&dBuffer, bufferSize);
    hipDeviceSynchronize();
    time_t mv2 = clock();
    checkSpError(hipsparseSpMV(handle, transA, alpha, matA_descr, vecX_descr, beta, vecY_descr, HIP_R_64F,
                              HIPSPARSE_SPMV_CSR_ALG2, dBuffer));
    hipDeviceSynchronize();
    cudaCheckError2();
    time_t mv3 = clock();
    hipsparseDestroySpMat(matA_descr);
    hipsparseDestroyDnVec(vecX_descr);
    hipsparseDestroyDnVec(vecY_descr);
    hipDeviceSynchronize();
    cudaCheckError2();
    time_t mv4 = clock();
    printf("mvTime %ld %ld %ld %ld \n",
           (mv4 - mv3) / (CLOCKS_PER_SEC / 1000), (mv3 - mv2) / (CLOCKS_PER_SEC / 1000),
           (mv2 - mv1) / (CLOCKS_PER_SEC / 1000),
           (mv1 - mv0) / (CLOCKS_PER_SEC / 1000));
}

void spSolverBiCGStab(int n, int nnz, const double *valA, const int *rowPtr, const int *colInd,
                      int *rowMap, int *colSortMap,
                      const double *b, double *x, double tol, hipsparseHandle_t cusparseHandle,
                      hipblasHandle_t cublasHandle) {
    time_t solve_start = clock();
    // Create descriptors for A, L and U.
    hipsparseMatDescr_t descrA, descrL, descrU;

    // Create ILU and SV info for A, L and U.
    csrilu02Info_t infoA;
    csrsv2Info_t infoL, infoU;

    // Create a copy of A for incomplete LU decomposition.
    // This copy will be modified in the solving process.
    double *valACopy;
    hipMalloc((void **) &valACopy, nnz * sizeof(double));
    hipMemcpy(valACopy, valA, nnz * sizeof(double), hipMemcpyDeviceToDevice);

    // Incomplete LU.
    time_t solve_start1 = clock();
    void *pBuffer;
    setUpMatrix(cusparseHandle, descrA, descrL, descrU, infoA, infoL, infoU, n, nnz, valACopy, rowPtr, colInd,
                &pBuffer);
    int *diag_info = nullptr;
    hipMalloc((void **) &diag_info, sizeof(int) * n);
    const int grid_size = ceil((double) n / (double) 32);
//    find_locn_of_diag_elements(n, diag_info, rowPtr,
//                               colInd);
    cudaCheckError2();

    ILU0_MEGA(rowPtr, colInd, valACopy, rowMap, colSortMap,
              n,
              nnz);
    hipDeviceSynchronize();
    time_t solve_start2 = clock();
    double *r;
    checkCudaError(hipMalloc((void **) &r, n * sizeof(double)));
    double *rw;
    hipMalloc((void **) &rw, n * sizeof(double));
    double *p;
    hipMalloc((void **) &p, n * sizeof(double));
    double *ph;
    hipMalloc((void **) &ph, n * sizeof(double));
    double *t;
    hipMalloc((void **) &t, n * sizeof(double));
    double *q;
    hipMalloc((void **) &q, n * sizeof(double));
    double *s;
    checkCudaError(hipMalloc((void **) &s, n * sizeof(double)));
    time_t solve_start3 = clock();

    double one = 1, nega_one = -1, zero = 0;
    double alpha, negalpha, beta, omega, nega_omega;
    double temp1, temp2;
    double rho = 0.0, rhop;
    double nrmr0;
    double nrmr;
    int niter = 0;

    // Initial guess x0 (all zeros here).
    hipblasDscal(cublasHandle, n, &zero, x, 1);
    // 1: compute the initial residual r = b - A * x0.
    spNewMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &nega_one, descrA, valA, rowPtr,
            colInd, x, &zero, r);
    cudaCheckError2();

    hipblasDaxpy(cublasHandle, n, &one, b, 1, r, 1);
    // 2: copy r into rw and p.
    hipblasDcopy(cublasHandle, n, r, 1, rw, 1);
    hipblasDcopy(cublasHandle, n, r, 1, p, 1);
    time_t solve_start4 = clock();

    hipblasDnrm2(cublasHandle, n, r, 1, &nrmr0);


    // Repeat until convergence.

    while (true) {
        printf("niter %d ms %ld\n", niter, clock() / (CLOCKS_PER_SEC / 1000));
        time_t it0 = clock();
        rhop = rho;
        hipblasDdot(cublasHandle, n, rw, 1, r, 1, &rho);

        if (niter > 0) {
            // 12
            beta = (rho / rhop) * (alpha / omega);

            // 13, p = r + beta * (p - omega * v)
            hipblasDaxpy(cublasHandle, n, &nega_omega, q, 1, p, 1);  // p += -omega * v
            hipblasDscal(cublasHandle, n, &beta, p, 1);  // p *= beta
            hipblasDaxpy(cublasHandle, n, &one, r, 1, p, 1);  // p += 1 * r
        }
        hipDeviceSynchronize();
        time_t it1 = clock();
//
//        int *myRow = new int[n + 1];
//        int *myCol = new int[nnz];
//        double *myVal = new double[nnz];
//        hipMemcpy(myRow, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
//        hipMemcpy(myCol, colInd, (nnz) * sizeof(int), hipMemcpyDeviceToHost);
//        hipMemcpy(myVal, valACopy, (nnz) * sizeof(double), hipMemcpyDeviceToHost);

//        double *myRhs = new double[n];
//        double *mySolve = new double[n];
//        hipMemcpy(myRhs, p, (n) * sizeof(double), hipMemcpyDeviceToHost);

        // 15: solve M * pw = p for pw.
//        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrL, valACopy, rowPtr,
//                              colInd, infoL, p, t, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, p, t, true);
        hipDeviceSynchronize();


        time_t it2 = clock();
//        double *myRhs = new double[n];
//        double *mySolve = new double[n];
//        hipMemcpy(myRhs, t, (n) * sizeof(double), hipMemcpyDeviceToHost);


//        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrU, valACopy, rowPtr,
//                              colInd, infoU, t, ph, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, t, ph, false);
//        hipMemcpy(mySolve, ph, (n) * sizeof(double), hipMemcpyDeviceToHost);
//        std::ofstream out3;
//        out3.open("/home/featurize/data/mock2.mtx");
//        for (int i = 0; i <= n; i++) {
//            out3 << myRow[i] << " ";
//        }
//        out3 << "\n";
//        for (int i = 0; i < nnz; i++) {
//            out3 << myCol[i] << " ";
//        }
//        out3 << "\n";
//        for (int i = 0; i < nnz; i++) {
//            out3 << std::setprecision(8) << myVal[i];
//            out3 << " ";
//        }
//        out3 << "\n";
//        for (int i = 0; i < n; i++) {
//            out3 << myRhs[i];
//            out3 << " ";
//        }
//        out3 << "\n";
//        for (int i = 0; i < n; i++) {
//            out3 << mySolve[i];
//            out3 << " ";
//        }
//        out3.flush();
//        throw 3;
        // 16
        spNewMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, valA, rowPtr, colInd,
                ph, &zero, q);
        hipDeviceSynchronize();
        time_t it3 = clock();
        // 17
        hipblasDdot(cublasHandle, n, rw, 1, q, 1, &temp1);
        alpha = rho / temp1;
        negalpha = -alpha;

        // 18
        hipblasDaxpy(cublasHandle, n, &negalpha, q, 1, r, 1);

        // 19
        hipblasDaxpy(cublasHandle, n, &alpha, ph, 1, x, 1);
        // 20
        hipblasDnrm2(cublasHandle, n, r, 1, &nrmr);
        hipDeviceSynchronize();
        time_t it4 = clock();
        if (nrmr / nrmr0 < tol) {
            std::cout << std::setprecision(12) << nrmr / nrmr0 << " NRMR \n";
            break;
        }

        // 23: solve M * sh = r for sh, note that s is sh for now.
//        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrL, valACopy, rowPtr,
//                              colInd, infoL, r, t, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, r, t, true);
        hipDeviceSynchronize();
        time_t it5 = clock();
//        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrU, valACopy, rowPtr,
//                              colInd, infoU, t, s, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
        spTrSolve(rowPtr, colInd, valACopy, n, nnz, t, s, false);


        // 24
        spNewMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, valA, rowPtr, colInd,
                s, &zero, t);


        // 25: omega = np.dot(t, r) / np.dot(t, t).
        hipblasDdot(cublasHandle, n, t, 1, r, 1, &temp1);
        hipDeviceSynchronize();
        time_t it6 = clock();
        hipblasDdot(cublasHandle, n, t, 1, t, 1, &temp2);

        omega = temp1 / temp2;
        nega_omega = -omega;

        // 26

        hipblasDaxpy(cublasHandle, n, &omega, s, 1, x, 1);

        hipblasDaxpy(cublasHandle, n, &nega_omega, t, 1, r, 1);

        hipblasDnrm2(cublasHandle, n, r, 1, &nrmr);
        hipDeviceSynchronize();
        time_t it7 = clock();
        printf("itTime %ld %ld %ld %ld %ld %ld %ld \n", (it7 - it6) / (CLOCKS_PER_SEC / 1000),
               (it6 - it5) / (CLOCKS_PER_SEC / 1000),
               (it5 - it4) / (CLOCKS_PER_SEC / 1000),
               (it4 - it3) / (CLOCKS_PER_SEC / 1000),
               (it3 - it2) / (CLOCKS_PER_SEC / 1000),
               (it2 - it1) / (CLOCKS_PER_SEC / 1000),
               (it1 - it0) / (CLOCKS_PER_SEC / 1000));
        if (nrmr / nrmr0 < tol) {
            std::cout << std::setprecision(12) << nrmr / nrmr0 << " NRMR \n";
            break;
        }

        niter++;

        //printf("Norm: %f\n", nrmr);
    }
    time_t solve_start5 = clock();

    //printf("Number of iterations: %d\n", niter);

    // Clean up
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyMatDescr(descrL);
    hipsparseDestroyMatDescr(descrU);
    hipsparseDestroyCsrilu02Info(infoA);
    hipsparseDestroyCsrsv2Info(infoL);
    hipsparseDestroyCsrsv2Info(infoU);
    hipFree(r);
    hipFree(rw);
    hipFree(p);
    hipFree(ph);
    hipFree(t);
    hipFree(q);
    hipFree(s);
    hipFree(valACopy);
    hipFree(pBuffer);;
    time_t solve_start6 = clock();
    time_t solve_end = clock();

    printf("solveTime %ld %ld %ld %ld %ld %ld %ld %ld\n", (solve_end - solve_start) / (CLOCKS_PER_SEC / 1000),
           (solve_end - solve_start6) / (CLOCKS_PER_SEC / 1000),
           (solve_start6 - solve_start5) / (CLOCKS_PER_SEC / 1000),
           (solve_start5 - solve_start4) / (CLOCKS_PER_SEC / 1000),
           (solve_start4 - solve_start3) / (CLOCKS_PER_SEC / 1000),
           (solve_start3 - solve_start2) / (CLOCKS_PER_SEC / 1000),
           (solve_start2 - solve_start1) / (CLOCKS_PER_SEC / 1000),
           (solve_start1 - solve_start) / (CLOCKS_PER_SEC / 1000));
}




